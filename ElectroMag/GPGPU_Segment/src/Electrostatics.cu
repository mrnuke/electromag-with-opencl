/***********************************************************************************************
Copyright (C) 2009-2010 - Alexandru Gagniuc - <http:\\g-tech.homeserver.com\HPC.htm>
 * This file is part of ElectroMag.

    ElectroMag is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ElectroMag is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with ElectroMag.  If not, see <http://www.gnu.org/licenses/>.
***********************************************************************************************/
/*////////////////////////////////////////////////////////////////////////////////
 See Makefile for compilation options
*/////////////////////////////////////////////////////////////////////////////////
#include "Electrostatics kernel.cuh"
#include "Electrostatics MT kernel.cuh"
#include "Config.h"


void templatizer(void)
{
    /*
     * The nvcc compiler will not compile kernels that are written as function
     * temlates. Therefore, we need to call each kernel template with the template
     * values that will be used in order for those kernels to be included in the
     * ptx code.
     * Although the call is done using the runtime API, while the rest of the
     * application utilizes the driver API, this function will never get compiled
     * into an object file, and will never be linked into the application,
     * therefore no conflict between the two APIs should appear
     */
    CalcField_MTkernel_CurvatureCompute<1><<<1, 1>>>(0, 0, 0, 0, 0, 0, 0, 0);
    CalcField_MTkernel<1><<<1, 1>>>(0, 0, 0, 0, 0, 0, 0, 0);
    CalcField_SPkernel<1><<<1, 1>>>(0, 0, 0, 0, 0, 0, 0, 0);
    CalcField_MTkernel_DP<1><<<1, 1>>>(0, 0, 0, 0, 0, 0, 0, 0);
    CalcField_DPkernel<1><<<1, 1>>>(0, 0, 0, 0, 0, 0, 0, 0);

}